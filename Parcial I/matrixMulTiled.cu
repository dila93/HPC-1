#include <hip/hip_runtime.h>
#include <ctime>
#include <iostream>

#define TILE_WIDTH 32
#define endl '\n'

__global__
void multKernelTiled(float *d_M, float *d_N, float *d_R, int width_M, int height, int width_N) {

}

__global__
void multKernel(float *d_M, float *d_N, float *d_R, int width_M, int height, int width_N) {
  int i = blockIdx.y * blockDim.y + threadIdx.y;
  int j = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < height and j < width_N) {
    int Pvalue = 0;
    for (int k = 0; k < width_M; k++) {
      Pvalue += d_M[i * width_M + k] * d_N[k * width_N + j];
    }
    d_R[i * width_N + j] = Pvalue;
  }
}

void mult(float *A, float *B, float *C, int width_A, int height_A, int width_B) {
  int aux = 0;
  for (int i = 0; i < height_A; i++) {
    for (int j = 0; j < width_B; j++) {
      aux = 0;
      for (int k = 0; k < width_A; k++)
        aux += A[i * width_A + k] * B[k * width_B + j];
      C[i * width_B + j] = aux;
    }
  }
}

void initValues(float *m, int width, int height) {
  int values = 1;
  for (int i = 0; i < height; i++) {
    for (int j = 0; j < width; j++) {
      m[i * width + j] = values++;
    }
  }
}

void print(float *m, int width, int height) {
  for (int i = 0; i < height; i++) {
    for (int j = 0; j < width; j++) {
      if (j) std::cout << " ";
      std::cout << m[i * width + j];
    }
    std::cout << endl;
  }
}

int main() {
  int height = 3;
  int width_A = 3;
  int width_B = 2;

  float *A = new float[height * width_A];
  float *B = new float[height * width_B];
  float *C = new float[height * width_B];
  float *D = new float[height * width_B];

  initValues(A, width_A, height);
  initValues(B,width_B, height);

  float *d_A, *d_B, *d_D;
  int blocksize = 32;

  dim3 dimBlock(blocksize, blocksize, 1);
  dim3 dimGrid(ceil(width_B / float(blocksize)), ceil(height / float(blocksize)), 1);

  hipMalloc((void**)&d_A, sizeof(float) * height * width_A);
  hipMalloc((void**)&d_B, sizeof(float) * height * width_B);
  hipMalloc((void**)&d_D, sizeof(float) * height * width_B);
  std::cout << std::fixed;

  {
    clock_t start = clock();
    mult(A, B, C, width_A, height, width_B);
    clock_t end = clock();
    double cpu_time_used = double(end - start) / CLOCKS_PER_SEC;
    std::cout << "Tiempo invertido CPU = " << cpu_time_used << "s\n";
    print(C, width_B, height);

  }

  // Mult without tiles
  {
    clock_t start = clock();

    hipMemcpy(d_A, A, sizeof(float) * height * width_A, hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, sizeof(float) * height * width_B, hipMemcpyHostToDevice);

    multKernel<<<dimGrid, dimBlock>>>(d_A, d_B, d_D, width_A, height, width_B);
    hipMemcpy(D, d_D, sizeof(float) * height * width_B, hipMemcpyDeviceToHost);

    clock_t end = clock();
    double cpu_time_used = double(end - start) / CLOCKS_PER_SEC;
    std::cout << "Tiempo invertido GPU = " << cpu_time_used << "s\n";
    print(D, width_B, height);
  }

  // Mult with tiles
  {
    clock_t start = clock();

    hipMemcpy(d_A, A, sizeof(float) * height * width_A, hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, sizeof(float) * height * width_B, hipMemcpyHostToDevice);

    multKernelTiled<<<dimGrid, dimBlock>>>(d_A, d_B, d_D, width_A, height, width_B);
    hipMemcpy(D, d_D, sizeof(float) * height * width_B, hipMemcpyDeviceToHost);

    clock_t end = clock();
    double cpu_time_used = double(end - start) / CLOCKS_PER_SEC;
    std::cout << "Tiempo invertido GPU = " << cpu_time_used << "s\n";
    print(D, width_B, height);
  }

  delete A;
  delete B;
  delete C;
  delete D;

  hipFree(d_A);
  hipFree(d_B);
  hipFree(d_D);

  return 0;
}